#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstring>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "functions.hpp"

using namespace std;

/**
 * Output for each state, as a bitwise mask.
 * Bit i in this mask is on if the keyword with index i appears when the machine
 * enters this state.
 */
int output[MAXS];

// Used internally in the algorithm.
int failure_function[MAXS]; // Failure function.
int goto_function[MAXS][MAXC]; // Goto function, -1 if fail.

__host__ int build_matching_machine(char **keywords, int keywords_size,
                            char lowest_char = 'a', char highest_char = 'z')
{
    memset(output, 0, sizeof output);
    memset(failure_function, -1, sizeof failure_function);
    memset(goto_function, -1, sizeof goto_function);

    int states = 1; // Initially, we just have the 0 state

    for(int i = 0; i < keywords_size; i++){
        const string keyword (keywords[i]);
        int current_state = 0;
        for(int j = 0; j < keyword.size(); j++){
            int c = keyword[j] - lowest_char;
            if(goto_function[current_state][c] == -1){ // Allocate a new node
                goto_function[current_state][c] = states++;
            }
            current_state = goto_function[current_state][c];
        }
        // There's a match of keywords[i] at node current_state.
        output[current_state] |= (1 << i);
    }

    // State 0 should have an outgoing edge for all characters.
    for(int c = 0; c < MAXC; c++){
        if(goto_function[0][c] == -1){
            goto_function[0][c] = 0;
        }
    }

    // Building the failure function
    queue_t *q = create_queue();
    data_u *dt = NULL;
    for(int c = 0; c <= highest_char - lowest_char; c++){
        // All nodes s of depth 1 have failure_function[s] = 0
        if(goto_function[0][c] != -1 and goto_function[0][c] != 0){
            failure_function[goto_function[0][c]] = 0;
            dt = (data_u *)malloc(sizeof(data_u));
            dt->state = goto_function[0][c];
            push(q, dt);
        }
    }
    while(q->size){
        dt = pop(q);
        int state = dt->state;
        free(dt);
        for(int c = 0; c <= highest_char - lowest_char; c++){
            if(goto_function[state][c] != -1){
                int failure = failure_function[state];
                while(goto_function[failure][c] == -1){
                    failure = failure_function[failure];
                }
                failure = goto_function[failure][c];
                failure_function[goto_function[state][c]] = failure;
                // Merge out values
                output[goto_function[state][c]] |= output[failure];
                dt = (data_u *)malloc(sizeof(data_u));
                dt->state = goto_function[state][c];
                push(q, dt);
            }
        }
    }
    free(q);

    return states;
}

__device__ int find_next_state(int current_state, char next_input,
		int* d_goto_function, int* d_failure_function, char lowest_char = 'a')
{
    int answer = current_state;
    int c = next_input - lowest_char;
    /*while(d_goto_function[answer][c] == -1)
        answer = d_failure_function[answer];*/
    while(d_goto_function[answer*MAXC + c] == -1)
            answer = d_failure_function[answer];
    //return d_goto_function[answer][c];
    return d_goto_function[answer*MAXC + c];
}

__global__ void search_pattern(location_t *d_locations, int keywords_size,
                                int *d_key_sizes, char *d_text, int num_lines,
                                int *d_line_pos, int* d_goto_function,
                                int* d_failure_function, int* d_output,
                                char lowest_char = 'a')
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int current_state = 0, loc = 0; // Ver concorrencia de threads no d_locations

    if(idx < num_lines){
    	for(int i = d_line_pos[idx]; d_text[i] != '\n'; i++){
			current_state = find_next_state(current_state, d_text[i],
                                            d_goto_function, d_failure_function,
                                            lowest_char);
			if(d_output[current_state] == 0)
				continue; // Nothing new, let's move on to the next character.
			for(int j = 0; j < keywords_size; j++){
				if (d_output[current_state] & (1 << j)) { // Matched keywords[j]
					d_locations[loc].keyword = j;
					d_locations[loc].begin = i - d_key_sizes[j] + 1;
					d_locations[loc].end = i;
					loc++;
				}
			}
		}
    }
}
