#include <iostream>
#include <fstream>
#include <string>
#include <cstdlib>
#include <cstring>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "functions.hpp"

//#define MAX_FILE_SIZE 1073741824 // Value in bytes. (1GB)
#define MAX_FILE_SIZE 1048576 // Value in bytes. (1MB)
#define MAX_LINE_SIZE 200
#define MAX_LINES 	  1000
#define MAX_LOCATIONS 500

using namespace std;

extern const int MAXS;
extern const int MAXC;
extern int failure_function[MAXS];
extern int goto_function[MAXS][MAXC];
extern int output[MAXS];

int main(int argc, char** argv)
{
    if(argc < 3){
        cout << "Usage:\n" << argv[0] << " <keywords file name>"
        << " <text file name>" << endl;
        cout << "Example:\n" << argv[0] << " keywords.txt input_text.txt"
        << endl;
        exit(1);
    }

    char **keywords;
    int keywords_size, total_length;

    cout << "Reading keywords..." << endl;
    keywords = read_keywords(argv[1], keywords_size, total_length);

    cout << "Keywords:" << endl;
    for(int i = 0; i < keywords_size; i++)
        cout << keywords[i] << endl;

    cout << "Building goto function..." << endl;
    //int total_states = build_matching_machine(keywords, keywords_size,'a', 'z');
    int total_states = build_matching_machine(keywords, keywords_size, '!', '~');
    if(total_states > 1)
        cout << "Goto built!" << endl;
    else
        cout << "Could not build goto function" << endl;

    cout << "\nMatching machine:\n";
    //print_matching_machine('a', 'z', total_states);
    print_matching_machine('!', '~', total_states);

    cout << "\nFailure function:\n";
    print_failure_function(total_states);

    fstream file;
    string text;
    char full_text[MAX_FILE_SIZE], new_line[MAX_LINE_SIZE];
    int key_sizes[keywords_size];

    // Reading input text file.
    for(int i = 0; i < keywords_size; i++)
        key_sizes[i] = strlen(keywords[i]);

    file.open(argv[2], fstream::in | fstream::out);

    int idx = 1, pos, line_pos[MAX_LINES], num_lines;
    full_text[0] = '\0';
	pos = num_lines = 0;
	line_pos[0] = pos;
    while(!file.fail() && !file.eof() && file.is_open()){
        getline(file, text);
        if(text.size() == 0)
        	continue;
        text.copy(new_line, text.size(), 0);
        new_line[text.size()] = '\0';
        strcat(full_text, new_line);
        pos += text.size() + 1;
        full_text[pos - 1] = '\n';
        full_text[pos] = '\0';
        line_pos[idx++] = pos;
        num_lines++;
    }

    file.close();

    // Making preparations for kernel launch.
    dim3 grid, block;
	//block.x = 1024;
	//grid.x = 15;
	block.x = 128;
	grid.x = 1;

	//location_t locations[MAX_LOCATIONS], *d_locations;
	location_t *locations, *d_locations;
	locations = (location_t *)malloc(MAX_LOCATIONS * sizeof(location_t));
	int *d_key_sizes, *d_line_pos;
	int *d_goto_function, *d_failure_function, *d_output;
	char *d_text;

	hipMalloc((void**)&d_locations, MAX_LOCATIONS * sizeof(location_t));
	hipMalloc((void**)&d_key_sizes, keywords_size * sizeof(int));
	hipMalloc((void**)&d_line_pos, MAX_LINES * sizeof(int));
	hipMalloc((void**)&d_goto_function, MAXS*MAXC * sizeof(int));
	hipMalloc((void**)&d_failure_function, MAXS * sizeof(int));
	hipMalloc((void**)&d_output, MAXS * sizeof(int));
	hipMalloc((void**)&d_text, MAX_FILE_SIZE * sizeof(char));

	hipMemcpy(d_key_sizes, key_sizes, keywords_size * sizeof(int),
                hipMemcpyHostToDevice);
	hipMemcpy(d_line_pos, line_pos, num_lines * sizeof(int),
                hipMemcpyHostToDevice);
	hipMemcpy(d_failure_function, failure_function, total_states * sizeof(int),
                hipMemcpyHostToDevice);
	hipMemcpy(d_output, output, MAXS * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_text, full_text, strlen(full_text) * sizeof(char),
                hipMemcpyHostToDevice);

	for(int i = 0; i < MAXS; i++)
		hipMemcpy(&d_goto_function[i*MAXC], goto_function[i],
                    MAXC * sizeof(char), hipMemcpyHostToDevice);

    cout << "\nSearching occurrences of keywords in the text...\n";
	//search_pattern(locations, keywords_size, key_sizes, text, 'a');
    search_pattern<<<grid, block>>>(d_locations, keywords_size, d_key_sizes,
                                    d_text, num_lines, d_line_pos,
                                    d_goto_function, d_failure_function,
                                    d_output, '!');

    hipMemcpy(locations, d_locations, MAX_LOCATIONS * sizeof(location_t),
                hipMemcpyDeviceToHost);

	/*int line = 1;

	if(locations->size == 0){
		line++;
		continue;
	}

	cout << "\nOccurrences in line " << line++ << endl;*/
    cout << "\nOccurrences:\n";
	for(int i = 0; i < 10 && locations[i].keyword < keywords_size; i++){
		cout << "Keyword " << keywords[locations[i].keyword] << " appears from "
		<< locations[i].begin << " to " << locations[i].end << endl;
	}

    for(int i = 0; i < keywords_size; i++)
        free(keywords[i]);
    free(keywords);
    free(locations);

    return(0);
}
